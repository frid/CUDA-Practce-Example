#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

//CUDA Runtime
#include <hip/hip_runtime.h>

//Helper function
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "cuPrintf.cu"

__global__ void printThreadID(void)
{
	int BlockID = blockIdx.x;
	int ThreadID = threadIdx.x;
	printf("Block id x : %d, Thread id x : %d\n", BlockID, ThreadID);

}

int main(int argc, char **argv)
{

	int DevID = findCudaDevice(argc, (const char**)argv);
	hipDeviceProp_t props;

	hipGetDevice(&DevID);
	hipGetDeviceProperties(&props, DevID);

	dim3 grid(3, 1, 1);     //Block size, SM size
	dim3 threads(10, 1, 1); //Thread size, SP size, cuda core size

	printThreadID<<<grid, threads>>>();
	hipDeviceSynchronize();

	hipDeviceReset();

	getchar();
	return EXIT_SUCCESS;
}

